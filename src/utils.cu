#include "utils.cuh"
#include <hiprand.h>

void GenerateRandomNumCuRand(float* data,int64_t nums,int64_t seed) {
    float *data_d;
    hipMalloc(&data_d,nums*sizeof(float));
    hiprandGenerator_t gen;
    hiprandCreateGenerator(&gen,HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen,seed);
    hiprandGenerateUniform(gen,data_d,nums);
    hipMemcpy(data,data_d,nums*sizeof(float),hipMemcpyDefault);
    hipFree(data_d);
    hiprandDestroyGenerator(gen);
}

void GenerateRandomIntegerCuRand(int64_t* data,int64_t nums,int64_t seed) {
    unsigned int* data_d;
    unsigned int *data1 = new unsigned int[nums];
    hipMalloc(&data_d,nums*sizeof(unsigned int));
    hiprandGenerator_t gen;
    hiprandCreateGenerator(&gen,HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen,seed);
    hiprandGeneratePoisson(gen,data_d,nums,seed);
    hipMemcpy(data1,data_d,nums*sizeof(unsigned int),hipMemcpyDefault);
    hipFree(data_d);
    hiprandDestroyGenerator(gen);
    for (int64_t i=0;i<nums;i++) data[i]=static_cast<int64_t>(data1[i]);
    free(data1);
}